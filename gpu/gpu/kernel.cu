#include "hip/hip_runtime.h"

#include <stdio.h>  
#include <stdlib.h>  
#include <hip/hip_runtime.h>
#include"Lock.h"
#include"random.h"
//#include <guiddef.h>
#include ""

//�쳣���
static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define TESTCPPDLL_API __declspec(dllexport)
#define CHECK(res) if(res!=hipSuccess){exit(-1);}
extern "C" TESTCPPDLL_API float* __stdcall Run(float* prices, int rows, int cols, bool savefile);

using namespace std;

//--�豸������ͬ����--//
static int selectGpuCount;						//ѡ���gpu����
static int *gpuDeviceID = NULL;					//ѡ���gpu���豸ID��
static int *gpuDeviceIsBusy = NULL;				//GPU�豸�Ƿ�æµ
static int StreamNum = 0;                       //3�״�������������
Mutex gpuDevLock;
CLock gpuLock(gpuDevLock);
//-------------------//

//��ʼ��CUDA�豸
void iniCUDA() {
	int gpuCount, i;
	int dev;
	HANDLE_ERROR(hipGetDeviceCount(&gpuCount));
	for (dev = 0; dev < gpuCount; dev++)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		StreamNum = deviceProp.multiProcessorCount*8;
	}


	if (gpuCount < 1)
	{
		fprintf(stderr, "There is no GPU device.");
		exit(1);
	}
	selectGpuCount = gpuCount;
	allocateMemory1D(&gpuDeviceID, selectGpuCount);
	allocateMemory1D(&gpuDeviceIsBusy, selectGpuCount);
	generateUniqueNNumbers(gpuDeviceID, selectGpuCount, 0, selectGpuCount - 1);
	memset(gpuDeviceIsBusy, 0, selectGpuCount * sizeof(*gpuDeviceIsBusy));
}
//DLL��װ����
BOOL APIENTRY DllMain(HANDLE hModule,
	DWORD  ul_reason_for_call,
	LPVOID lpReserved
)
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH: {
		//��ʼCUDA
		iniCUDA();
	}
	case DLL_THREAD_ATTACH:
	case DLL_THREAD_DETACH:
	case DLL_PROCESS_DETACH:
		break;
	}
	return TRUE;
}

__global__ void tensor(float **da, float ** preda, long rows, long  cols, float *D, int ite)
{
	long row = blockDim.y*blockIdx.y + threadIdx.y;
	long col = blockDim.x*blockIdx.x + threadIdx.x;

	if (row < rows && row>(2 * cols) && col > 1 && col < cols + 1)
		if (ite < col)
		{
			float re = 0;
			float rate = 0;
			for (int j = 0; j < col; j++)
			{
				int pos = j + ite;
				re += preda[row - pos][0];
			}
			rate = (da[row - ite][0] - da[row - ite - col][0]) / da[row - ite - col][0];
			//�����豸֧��ԭ�Ӳ������ɽ���3�׶ε������в���
			atomicAdd(&da[row][col - 1], 100000000 * (re - rate));
			//da[row][col - 1] += 100000000 * (re - rate);
		}
}

__global__ void tensor_data(float **da, unsigned int rows, unsigned int cols, const float *D)
{
	unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;
	//unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;
	if (row < rows)
		if (row > 1)
			da[row][0] = (D[row] - D[row - 1]) / D[row - 1];
		else
			da[row][0] = D[row];
}

__global__ void copypricedata(float **da, unsigned int rows, unsigned int cols, const float *D)
{
	unsigned int row = blockDim.y*blockIdx.y + threadIdx.y;
	//unsigned int col = blockDim.x*blockIdx.x + threadIdx.x;
	if (row < rows)
		da[row][0] = D[row];
}

//�ں˼���
float* KernelComputer(float* prices, int rows, int cols, bool savefile)
{
	printf("[tensor of  %d elements]\n", rows);
	hipError_t err = hipSuccess;
	hipDeviceReset();
	size_t size = rows * sizeof(float);
	float *h_close = (float *)malloc(size);
	for (int i = 0; i < rows; i++)
		h_close[i] = prices[i];
	float *d_close = NULL;
	hipMalloc((void **)&d_close, size);
	hipMemcpy(d_close, h_close, size, hipMemcpyHostToDevice);

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = (rows + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	//--1,2�������в���--//
	//1��
	static float **da = NULL;
	float **ha = NULL;
	float *dc = NULL;
	float *hc = NULL;
	//2��
	float **da1 = NULL;
	float **ha1 = NULL;
	float *dc1 = NULL;
	//------------------//
	hipError_t res;
	//--����׼��--//
	//1��
	int r, c;
	res = hipMalloc((void**)(&da), rows * sizeof(float*)); CHECK(res);
	res = hipMalloc((void**)(&dc), rows*cols * sizeof(float)); CHECK(res);

	ha = (float**)malloc(rows * sizeof(float*));
	hc = (float*)malloc(rows*cols * sizeof(float));

	for (r = 0; r < rows; r++)
		ha[r] = dc + r*cols;

	dim3 dimBlock(32, 32);
	dim3 dimGrid((cols + dimBlock.x - 1) / (dimBlock.x), (rows + dimBlock.y - 1) / (dimBlock.y));
	//2��
	res = hipMalloc((void**)(&da1), rows * sizeof(float*)); CHECK(res);
	res = hipMalloc((void**)(&dc1), rows*cols * sizeof(float)); CHECK(res);
	ha1 = (float**)malloc(rows * sizeof(float*));
	for (r = 0; r < rows; r++)
		ha1[r] = dc1 + r*cols;

	printf("preparing data...\n");
	//-------------------//
	//--1,2�����в���--//
    //��������������
	hipStream_t stream[2];
	for (int i = 0; i < 2; i++) {
		hipStreamCreate(&stream[i]);
	}
	//1��
	res = hipMemcpyAsync((void*)(da), (void*)(ha), rows * sizeof(float*), hipMemcpyHostToDevice,stream[0]); CHECK(res);
	copypricedata << <dimGrid, dimBlock,0,stream[0] >> > (da, rows, cols, d_close);
	//2��
	res = hipMemcpyAsync((void*)(da1), (void*)(ha1), rows * sizeof(float*), hipMemcpyHostToDevice,stream[1]); CHECK(res);
	tensor_data << <dimGrid, dimBlock,0,stream[1] >> > (da1, rows, cols, d_close);
	//-----------------//
	//ͬ��(����������Դ��Ҫ�ⲿ�ֽ��)
	res = hipDeviceSynchronize();
	if (res != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", res);
		exit(EXIT_FAILURE);
	}
	//������
	for (int i = 0; i < 2; i++) {
		hipStreamDestroy(stream[i]);
	}
	//3�׼���
	hipStream_t *streamT=new hipStream_t[StreamNum];
	for (int i = 0; i < StreamNum; i++) {
		hipStreamCreate(&streamT[i]);
	}
	printf("computing tensor..\n");
	for (int ite = 0; ite < cols; ++ite)
	{
		tensor << <dimGrid, dimBlock,0,streamT[ite%StreamNum]>> > (da, da1, rows, cols, d_close, ite);
		if((ite%StreamNum)==(StreamNum-1))hipDeviceSynchronize();
	}
	res = hipMemcpy((void*)(hc), (void*)(dc), rows*cols * sizeof(float), hipMemcpyDeviceToHost); CHECK(res);

	if (savefile) {
		printf("saving to file...\n");
		FILE * output = fopen("output-gpu.txt", "w");
		for (r = 0; r < rows; r++)
		{
			for (c = 0; c < cols; c++)
				fprintf(output, "%f\t", hc[r*cols + c]);
			fprintf(output, "\n");
		}
		fclose(output);
	}
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	//������
	for (int i = 0; i < StreamNum; i++) {
		hipStreamDestroy(streamT[i]);
	}
	delete[] streamT;
	hipFree(d_close);
	hipFree((void*)da);
	hipFree((void*)dc);
	free(h_close);
	free(ha);
	printf("ok...\n");
	return hc;
}

TESTCPPDLL_API float* __stdcall Run(float* prices, int rows, int cols, bool savefile)
{
	//���ָ��
	float *runPort = NULL;
	int i = 0;
	int aVacantGpu;
	bool retry = true;
	while (retry)
	{
		if (getARandom(0, 128) % 2 == 0)
		{
			gpuLock.Lock();
			for (i = 0; i < selectGpuCount; i++)
			{
				if (!gpuDeviceIsBusy[gpuDeviceID[i]])	//�ҵ�һ�����е�gpu�豸
				{
					aVacantGpu = gpuDeviceID[i];
					gpuDeviceIsBusy[gpuDeviceID[i]] = 1;//����ռ�ñ�־�������豸״̬Ϊæ��

					gpuLock.Unlock();					//�ǵü�ʱ�ͷ���
					retry = false;

					//printf("Thread have get a vacant gpu device%d.\n", aVacantGpu);
					break;
				}
			}
			if (i >= selectGpuCount)					//����gpu�豸����ռ����
			{
				gpuLock.Unlock();						//��ʱ�ͷ���

														//printf("No gpu device is vacant, thread  have to wait.\n");

				//Sleep(1);							//˯��һ����������
				retry = true;
			}
		}
		else
		{
			gpuLock.Lock();
			for (i = selectGpuCount - 1; i >= 0; i--)
			{
				if (!gpuDeviceIsBusy[gpuDeviceID[i]])	//�ҵ�һ�����е�gpu�豸
				{
					aVacantGpu = gpuDeviceID[i];
					gpuDeviceIsBusy[gpuDeviceID[i]] = 1;//����ռ�ñ�־�������豸״̬Ϊæ��

					gpuLock.Unlock();					//�ǵü�ʱ�ͷ���
					retry = false;

					//printf("Thread have get a vacant gpu device%d.\n", aVacantGpu);
					break;
				}
			}
			if (i < 0)									//����gpu�豸����ռ����
			{
				gpuLock.Unlock();						//��ʱ�ͷ���

				//Sleep(1);							//˯��һ����������
				retry = true;

				//printf("No gpu device is vacant, thread have to wait.\n");
			}
		}
	}
	hipSetDevice(aVacantGpu);
	//�ں˼���
	runPort=KernelComputer(prices,rows,cols,savefile);
	//�ǵü�ʱ�黹gpu�����ÿ��б�־Ҳ��Ҫͬ������Ϊ�����������̶߳�ȡ
	//gpuLockһ����������gpuDeviceIsBusy
	gpuLock.Lock();
	gpuDeviceIsBusy[aVacantGpu] = 0;
	gpuLock.Unlock();
	return runPort;
}